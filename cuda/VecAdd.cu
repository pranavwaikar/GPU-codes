#include "hip/hip_runtime.h"
// headers
#include <stdio.h>

#include <hip/hip_runtime.h> // for CUDA

#include "helper_timer.h"

// global variables
// odd number 11444777 is deliberate illustration ( Nvidia OpenCL Samples )
int iNumberOfArrayElements=5;

float *hostInput1=NULL;
float *hostInput2=NULL;
float *hostOutput=NULL;
float *gold=NULL;

float *deviceInput1=NULL;
float *deviceInput2=NULL;
float *deviceOutput=NULL;

float timeOnCPU;
float timeOnGPU;

// *** CUDA KERNEL DEFINITION ***
// global kernel function definition
__global__ void vecAdd(float *in1,float *in2,float *out,int len)
{
    // variable declarations
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    // code
    if(i < len)
    {
        out[i]=in1[i]+in2[i];
    }
}

int main(int argc,char *argv[])
{
    // function declarations
    void fillFloatArrayWithRandomNumbers(float *, int);
    void vecAddHost(const float *, const float *, float *, int);
    void cleanup(void);
    
    // code
    // allocate host-memory
    hostInput1=(float *)malloc(sizeof(float) * iNumberOfArrayElements);
    if(hostInput1== NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 1.\nExitting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    hostInput2=(float *)malloc(sizeof(float) * iNumberOfArrayElements);
    if(hostInput2== NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 2.\nExitting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    hostOutput=(float *)malloc(sizeof(float) * iNumberOfArrayElements);
    if(hostOutput== NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Output Array.\nExitting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    gold=(float *)malloc(sizeof(float) * iNumberOfArrayElements);
    if(gold== NULL)
    {
        printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Gold Output Array.\nExitting ...\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // fill above input host vectors with arbitary but hard-coded data
    fillFloatArrayWithRandomNumbers(hostInput1,iNumberOfArrayElements);
    fillFloatArrayWithRandomNumbers(hostInput2,iNumberOfArrayElements);
    
    // allocate device-memory
    hipError_t err=hipSuccess;
    err=hipMalloc((void **)&deviceInput1,sizeof(float) * iNumberOfArrayElements);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    err=hipMalloc((void **)&deviceInput2,sizeof(float) * iNumberOfArrayElements);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    err=hipMalloc((void **)&deviceOutput,sizeof(float) * iNumberOfArrayElements);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // copy host memory contents to device memory
    err=hipMemcpy(deviceInput1,hostInput1,sizeof(float) * iNumberOfArrayElements,hipMemcpyHostToDevice);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    err=hipMemcpy(deviceInput2,hostInput2,sizeof(float) * iNumberOfArrayElements,hipMemcpyHostToDevice);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // cuda kernel configuration
    dim3 DimGrid=dim3(ceil(iNumberOfArrayElements/256.0),1,1);
    dim3 DimBlock=dim3(256,1,1);
    
    // start timer
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    
    vecAdd<<<DimGrid,DimBlock>>>(deviceInput1,deviceInput2,deviceOutput,iNumberOfArrayElements);
    
    // stop timer
    sdkStopTimer(&timer);
    timeOnGPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    // copy device memory to host memory
    err=hipMemcpy(hostOutput,deviceOutput,sizeof(float) * iNumberOfArrayElements,hipMemcpyDeviceToHost);
    if(err!=hipSuccess)
    {
        printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n",hipGetErrorString(err),__FILE__,__LINE__);
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // results
    vecAddHost(hostInput1, hostInput2, gold, iNumberOfArrayElements);
    
    // compare results for golden-host
    const float epsilon = 0.000001f;
    bool bAccuracy=true;
    int breakValue=0;
    int i;
    for(i=0;i<iNumberOfArrayElements;i++)
    {
        float val1 = gold[i];
        float val2 = hostOutput[i];
        if(fabs(val1-val2) > epsilon)
        {
            bAccuracy = false;
            breakValue=i;
            break;
        }
    }
    
    if(bAccuracy==false)
    {
        printf("Break Value = %d\n",breakValue);
    }
    
    char str[125];
    if(bAccuracy==true)
        sprintf(str,"%s","Comparison Of Output Arrays On CPU And GPU Are Accurate Within The Limit Of 0.000001");
    else
        sprintf(str,"%s","Not All Comparison Of Output Arrays On CPU And GPU Are Accurate Within The Limit Of 0.000001");
    
    printf("1st Array Is From 0th Element %.6f To %dth Element %.6f\n",hostInput1[0], iNumberOfArrayElements-1, hostInput1[iNumberOfArrayElements-1]);
    printf("2nd Array Is From 0th Element %.6f To %dth Element %.6f\n",hostInput2[0], iNumberOfArrayElements-1, hostInput2[iNumberOfArrayElements-1]);
    printf("Grid Dimension = (%d,1,1) And Block Dimension = (%d,1,1)\n",DimGrid.x,DimBlock.x);
    printf("Sum Of Each Element From Above 2 Arrays Creates 3rd Array As :\n");
    printf("3nd Array Is From 0th Element %.6f To %dth Element %.6f\n",hostOutput[0], iNumberOfArrayElements-1, hostOutput[iNumberOfArrayElements-1]);
    printf("The Time Taken To Do Above Addition On CPU = %.6f (ms)\n",timeOnCPU);
    printf("The Time Taken To Do Above Addition On GPU = %.6f (ms)\n",timeOnGPU);
    printf("%s\n",str);
    
    // total cleanup
    cleanup();
    return(0);
}

void cleanup(void)
{
    // code
    
    // free allocated device-memory
    if(deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1=NULL;
    }
    
    if(deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2=NULL;
    }
    
    if(deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput=NULL;
    }
    
    // free allocated host-memory
    if(hostInput1)
    {
        free(hostInput1);
        hostInput1=NULL;
    }
    
    if(hostInput2)
    {
        free(hostInput2);
        hostInput2=NULL;
    }
    
    if(hostOutput)
    {
        free(hostOutput);
        hostOutput=NULL;
    }
    
    if(gold)
    {
        free(gold);
        gold=NULL;
    }
}

void fillFloatArrayWithRandomNumbers(float *pFloatArray, int iSize)
{
    // code
    int i;
    const float fScale = 1.0f / (float)RAND_MAX;
    for (i = 0; i < iSize; ++i)
    {
        pFloatArray[i] = fScale * rand();
    }
}

// "Golden" Host processing vector addition function for comparison purposes
void vecAddHost(const float* pFloatData1, const float* pFloatData2, float* pFloatResult, int iNumElements)
{
    int i;
    
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    
    for (i = 0; i < iNumElements; i++)
    {
        pFloatResult[i] = pFloatData1[i] + pFloatData2[i];
    }
    
    sdkStopTimer(&timer);
    timeOnCPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
}
